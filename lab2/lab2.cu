#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define CSC(call)  									                \
do {											                    \
	hipError_t res = call;							                \
	if (res != hipSuccess) {							            \
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);								                    \
	}										                        \
} while(0)

__global__ void kernel(hipTextureObject_t tex, uchar4* out, int w, int h, int wn, int hn) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy = blockDim.y * blockIdx.y + threadIdx.y;
  int offsetx = blockDim.x * gridDim.x;
  int offsety = blockDim.y * gridDim.y;

  int step_x = w / wn;
  int step_y = h / hn;
  int size = step_x * step_y;

  for (int y = idy; y < hn; y += offsety) {
    for (int x = idx; x < wn; x += offsetx) {

      float4 blockAcc = make_float4(0, 0, 0, 0);

      for (int j = 0; j < step_y; ++j) {
        for (int i = 0; i < step_x; ++i) {
          uchar4 p = tex2D<uchar4>(tex, (x * step_x + i + 0.5f) / w, (y * step_y + j + 0.5f) / h);
          blockAcc.x += p.x;
          blockAcc.y += p.y;
          blockAcc.z += p.z;
          blockAcc.w += p.w;
        }
      }

      out[y * wn + x] = make_uchar4(blockAcc.x / size, blockAcc.y / size, blockAcc.z / size, blockAcc.w / size);
    }
  }
}

int main() {
  char input_file[256], output_file[256];
  int wn, hn;

  scanf("%s", input_file);
  scanf("%s", output_file);
  scanf("%d %d", &wn, &hn);

  int w, h;
  FILE* fp = fopen(input_file, "rb");
  fread(&w, sizeof(int), 1, fp);
  fread(&h, sizeof(int), 1, fp);
  uchar4* data = (uchar4*)malloc(sizeof(uchar4) * w * h);
  fread(data, sizeof(uchar4), w * h, fp);
  fclose(fp);

  hipArray* arr;
  hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
  CSC(hipMallocArray(&arr, &ch, w, h));
  CSC(hipMemcpy2DToArray(arr, 0, 0, data, w * sizeof(uchar4), w * sizeof(uchar4), h, hipMemcpyHostToDevice));

  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = arr;

  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = true;

  hipTextureObject_t tex = 0;
  CSC(hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL));

  uchar4* dev_out;
  CSC(hipMalloc(&dev_out, sizeof(uchar4) * wn * hn));

  kernel << < dim3(16, 16), dim3(32, 32) >> > (tex, dev_out, w, h, wn, hn);
  CSC(hipGetLastError());

  uchar4* output_data = (uchar4*)malloc(sizeof(uchar4) * wn * hn);
  CSC(hipMemcpy(output_data, dev_out, sizeof(uchar4) * wn * hn, hipMemcpyDeviceToHost));

  CSC(hipDestroyTextureObject(tex));
  CSC(hipFreeArray(arr));
  CSC(hipFree(dev_out));

  fp = fopen(output_file, "wb");
  fwrite(&wn, sizeof(int), 1, fp);
  fwrite(&hn, sizeof(int), 1, fp);
  fwrite(output_data, sizeof(uchar4), wn * hn, fp);
  fclose(fp);

  free(data);
  free(output_data);
  return 0;
}
