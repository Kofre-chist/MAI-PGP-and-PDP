
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CSC(call)       \
do {                    \
    hipError_t status = call;          \
    if  (status != hipSuccess) {       \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status));   \
        exit(0);                        \
    }                                   \
} while (0)

__global__ void kernel(double *vec1, double *vec2, double *result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    while (idx < n) {
        result[idx] = vec1[idx] > vec2[idx] ? vec1[idx] : vec2[idx];
        idx += offset;
    }
}

int main() {
    long int n;
    scanf("%ld", &n);

    double *vec1 = (double *)malloc(sizeof(double) * n);
    double *vec2 = (double *)malloc(sizeof(double) * n);
    double *result = (double *)malloc(sizeof(double) * n);
    
    for (int i = 0; i < n; ++i){
      scanf("%lf", &vec1[i]);
    }

    for (int i = 0; i < n; ++i){
      scanf("%lf", &vec2[i]);
    }

    double *dev_vec1, *dev_vec2, *dev_result;

    CSC(hipMalloc(&dev_vec1, sizeof(double) * n));
    CSC(hipMalloc(&dev_vec2, sizeof(double) * n));
    CSC(hipMalloc(&dev_result, sizeof(double) * n));
    CSC(hipMemcpy(dev_vec1, vec1, sizeof(double) * n, hipMemcpyHostToDevice));
    CSC(hipMemcpy(dev_vec2, vec2, sizeof(double) * n, hipMemcpyHostToDevice));
    CSC(hipMemcpy(dev_result, result, sizeof(double) * n, hipMemcpyHostToDevice));

    kernel<<<1024, 1024>>>(dev_vec1, dev_vec2, dev_result, n);

    CSC(hipMemcpy(result, dev_result, sizeof(double) * n, hipMemcpyDeviceToHost));

    for (int i = 0; i < n; i++) {
        printf("%.10lf ", result[i]);
    }
    printf("\n");

    free(vec1);
    free(vec2);
    free(result);
    CSC(hipFree(dev_vec1));
    CSC(hipFree(dev_vec2));
    CSC(hipFree(dev_result));

    return 0;
}
